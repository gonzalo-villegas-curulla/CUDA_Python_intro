#include <iostream>
#include <math.h>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__
void add(int n, float *x, float *y){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x + gridDim.x;
    //if (idx<n){
        for (int IDX = idx; IDX<n; IDX += stride){
            y[IDX] = x[IDX] + y[IDX];
        }
    //}
}


int main(void){

    int N = 80;//1<<20;
    printf("      N = %d\n", N);
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // inits
    float val1 = 1.0, val2 = 2.0;

    for (int idx=0; idx<N; idx++){
        x[idx] = val1;
        y[idx] = val2;
    }

    int blocksize = 128;
    int gridsize = (N + blocksize - 1)/blocksize;

    std::cout << "Gsize: " << gridsize << " blocks. Bsize: " << blocksize << " threads per block." << std::endl; 
    add<<<gridsize, blocksize>>>(N, x, y);

    //handle kernel cuda errors

    //any error after synching CPU and GPU?
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess){
        std::cerr << "cuda error after synch: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // errorsGeorge in the kernel launch?
    err = hipGetLastError();
    if (err != hipSuccess){
        std::cerr << "cuda kernel launch error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }


    // value error check
    float maxErr = 0.0f;
    float total = val1+val2;
    for (int idx=0; idx<N; idx++){
        maxErr = fmax(maxErr, fabs(y[idx]- total));
    }
    std::cout << "MaxError: " << maxErr << std::endl;

    return 0;
}
