#include <iostream>
#include <math.h>
#include <cstdio>
#include <hip/hip_runtime.h>

__global__
void add(int n, float *x, float *y){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx<n){
        y[idx] = x[idx] + y[idx];
    }
}


int main(void){

    int N = 1<<30;
    float *x, *y;

    hipMallocManaged(&x, N*sizeof(float));
    hipMallocManaged(&y, N*sizeof(float));

    // inits
    float val1 = 1.0, val2 = 2.0;
    for (int idx=0; idx<N; idx++){
        x[idx] = val1;
        y[idx] = val2;
    }

    int blocksize =1024;
    int gridsize = (N + blocksize - 1)/blocksize;

    std::cout << "Gsize: " << gridsize << " blocks. Bsize: " << blocksize << " threads per block." << std::endl; 
    add<<<gridsize, blocksize>>>(N, x, y);

    //handle kernel cuda errors

    //any error after synching CPU and GPU?
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess){
        std::cerr << "cuda error after synch: " << hipGetErrorString(err) << std::endl;
        return -1;
    }

    // errorsGeorge in the kernel launch?
    err = hipGetLastError();
    if (err != hipSuccess){
        std::cerr << "cuda kernel launch error: " << hipGetErrorString(err) << std::endl;
        return -1;
    }


    // value error check
    float maxErr = 0.0f;
    float total = val1+val2;
    for (int idx=0; idx<N; idx++){
        maxErr = fmax(maxErr, fabs(y[idx]- total));
    }
    std::cout << "MaxError: " << maxErr << std::endl;

    return 0;
}
